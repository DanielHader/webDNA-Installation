#include "hip/hip_runtime.h"
#include "MD_SP_CUDAMPIBackend.h"
#include "CUDA_MD.cuh"
#include "../CUDA_base_interactions.h"
#include "../../Interactions/DNAInteraction.h"
#include "../Thermostats/CUDAThermostatFactory.h"
#include <thrust/sort.h>
#include <typeinfo>
#include <type_traits>

#include "../../Forces/COMForce.h"
#include "../../Forces/ConstantRateForce.h"
#include "../../Forces/ConstantRateTorque.h"
#include "../../Forces/ConstantTrap.h"
#include "../../Forces/LowdimMovingTrap.h"
#include "../../Forces/MovingTrap.h"
#include "../../Forces/MutualTrap.h"
#include "../../Forces/RepulsionPlane.h"
#include "../../Forces/RepulsionPlaneMoving.h"
#include "../../Forces/RepulsiveSphere.h"

__constant__ int verlet_N[1];

template <typename number, typename number4>
__global__ void check_partition(number4 *poss, int *part, int N,
                                number4 part_center, number4 part_radius,
                                number box_side, number boundary_size)
{
    if (IND >= N) return;
    
    number4 r = poss[IND];
    r.x = fmod(fmod(r.x, box_side) + box_side, box_side);
    r.y = fmod(fmod(r.y, box_side) + box_side, box_side);
    r.z = fmod(fmod(r.z, box_side) + box_side, box_side);

    number4 delta;
    delta.x = fabs(r.x - part_center.x);
    delta.y = fabs(r.y - part_center.y);
    delta.z = fabs(r.z - part_center.z);

    number4 inv_delta;
    inv_delta.x = box_side - delta.x;
    inv_delta.y = box_side - delta.y;
    inv_delta.z = box_side - delta.z;

    number4 min_delta;
    min_delta.x = fmin(delta.x, inv_delta.x);
    min_delta.y = fmin(delta.y, inv_delta.y);
    min_delta.z = fmin(delta.z, inv_delta.z);
    
    int contains = 0;

    if (min_delta.x <= part_radius.x and
        min_delta.y <= part_radius.y and
        min_delta.z <= part_radius.z)
    {
        contains = 1;
    }

    else if (min_delta.x <= part_radius.x + boundary_size and
        min_delta.y <= part_radius.y + boundary_size and
        min_delta.z <= part_radius.z + boundary_size)
    {
        contains = 2;
    }

    part[IND] = contains;
}

template <typename number, typename number4>
__global__ void redistribute_particles(number4 *poss, number4 *vels, number4 *Ls,
                                       GPU_quat<number> *orients, number4 *poss_buff,
                                       number4 *vels_buff, number4 *Ls_buff,
                                       GPU_quat<number> *orients_buff, int N)
{
    if (IND >= N) return;

    number4 r = poss_buff[IND];
    int idx = __float_as_int(r.w);

    poss[idx] = r;
    vels[idx] = vels_buff[IND];
    Ls[idx] = Ls_buff[IND];
    orients[idx] = orients_buff[IND];
}

template<typename number, typename number4>
MD_SP_CUDAMPIBackend<number, number4>::MD_SP_CUDAMPIBackend() : MD_CUDABackend<number, number4>() {
	this->_is_CUDA_sim = true;

    _h_partition = _d_partition = NULL;
    _h_buff_vels = _h_buff_Ls = NULL;
    _h_comp_poss = _h_comp_vels = NULL;
    _d_comp_poss = _d_comp_vels = NULL;
    _h_comp_Ls = _d_comp_Ls = NULL;
    _h_comp_orientations = _d_comp_orientations = NULL;
    _h_buff_orientations = _d_buff_orientations = NULL;
}

template<typename number, typename number4>
MD_SP_CUDAMPIBackend<number, number4>::~MD_SP_CUDAMPIBackend() {
    
    delete[] _h_partition;
    
    delete[] _h_buff_poss;
    delete[] _h_buff_vels;
    delete[] _h_buff_Ls;

    delete[] _h_comp_poss;
    delete[] _h_comp_vels;
    delete[] _h_comp_Ls;
    
    delete[] _h_buff_orientations;
    delete[] _h_comp_orientations;

    CUDA_SAFE_CALL( hipFree(_d_partition) );
    
    CUDA_SAFE_CALL( hipFree(_d_comp_poss) );
    CUDA_SAFE_CALL( hipFree(_d_comp_vels) );
    CUDA_SAFE_CALL( hipFree(_d_comp_Ls) );
    CUDA_SAFE_CALL( hipFree(_d_buff_poss) );
    CUDA_SAFE_CALL( hipFree(_d_buff_vels) );
    CUDA_SAFE_CALL( hipFree(_d_buff_Ls) );

    CUDA_SAFE_CALL( hipFree(_d_buff_orientations) );
    CUDA_SAFE_CALL( hipFree(_d_comp_orientations) );
}

template<typename number4>
void _init_mpi_type(MPI_Datatype &number4_type, MPI_Datatype &quat_type)
{
    if (std::is_same<number4, float4>::value)
    {
        int lengths[1] = {4};
        const MPI_Aint disps[1] = {0};
        MPI_Datatype types[1] = {MPI_FLOAT};

        MPI_Type_create_struct(1, lengths, disps, types, &number4_type);
        MPI_Type_commit(&number4_type);

        MPI_Type_create_struct(1, lengths, disps, types, &quat_type);
        MPI_Type_commit(&quat_type);
    }
    else
    {
        int lengths[2] = {3, 1};
        const MPI_Aint disps[2] = {0, sizeof(double) * 3};
        MPI_Datatype types[2] = {MPI_DOUBLE, MPI_FLOAT};

        MPI_Type_create_struct(2, lengths, disps, types, &number4_type);
        MPI_Type_commit(&number4_type);

        int qlengths[1] = {4};
        const MPI_Aint qdisps[1] = {0};
        MPI_Datatype qtypes[1] = {MPI_DOUBLE};

        MPI_Type_create_struct(1, qlengths, qdisps, qtypes, &quat_type);
        MPI_Type_commit(&quat_type);
    }
}

template<typename number, typename number4>
void MD_SP_CUDAMPIBackend<number, number4>::_compactify_partition()
{
    thrust::device_ptr<int> _d_partition_ptr(_d_partition);

    thrust::device_ptr<number4> _d_poss_ptr(this->_d_poss);
	thrust::device_ptr<number4> _d_vels_ptr(this->_d_vels);
    thrust::device_ptr<number4> _d_Ls_ptr(this->_d_Ls);
    thrust::device_ptr<GPU_quat<number> > _d_orientations_ptr(this->_d_orientations);
    
    thrust::device_ptr<number4> _d_poss_comp_ptr(this->_d_comp_poss);
	thrust::device_ptr<number4> _d_vels_comp_ptr(this->_d_comp_vels);
    thrust::device_ptr<number4> _d_Ls_comp_ptr(this->_d_comp_Ls);
    thrust::device_ptr<GPU_quat<number> > _d_orientations_comp_ptr(this->_d_comp_orientations);
    
    thrust::device_vector<int> _d_part_vec(this->_N);

    auto poss_tup_beg = thrust::make_zip_iterator(
        thrust::make_tuple(_d_partition_ptr, _d_poss_ptr));
    auto vels_tup_beg = thrust::make_zip_iterator(
        thrust::make_tuple(_d_partition_ptr, _d_vels_ptr));
    auto Ls_tup_beg = thrust::make_zip_iterator(
        thrust::make_tuple(_d_partition_ptr, _d_Ls_ptr));
    auto orientations_tup_beg = thrust::make_zip_iterator(
        thrust::make_tuple(_d_partition_ptr, _d_orientations_ptr));
    
    auto poss_tup_end = thrust::make_zip_iterator(
        thrust::make_tuple(_d_partition_ptr + this->_N, _d_poss_ptr + this->_N));
    auto vels_tup_end = thrust::make_zip_iterator(
        thrust::make_tuple(_d_partition_ptr + this->_N, _d_vels_ptr + this->_N));
    auto Ls_tup_end = thrust::make_zip_iterator(
        thrust::make_tuple(_d_partition_ptr + this->_N, _d_Ls_ptr + this->_N));
    auto orientations_tup_end = thrust::make_zip_iterator(
        thrust::make_tuple(_d_partition_ptr + this->_N, _d_orientations_ptr + this->_N));

    auto poss_comp_tup_beg = thrust::make_zip_iterator(
        thrust::make_tuple(_d_part_vec.begin(), _d_poss_comp_ptr));
    auto vels_comp_tup_beg = thrust::make_zip_iterator(
        thrust::make_tuple(_d_part_vec.begin(), _d_vels_comp_ptr));
    auto Ls_comp_tup_beg = thrust::make_zip_iterator(
        thrust::make_tuple(_d_part_vec.begin(), _d_Ls_comp_ptr));
    auto orientations_comp_tup_beg = thrust::make_zip_iterator(
        thrust::make_tuple(_d_part_vec.begin(), _d_orientations_comp_ptr));

    thrust::copy_if(poss_tup_beg, poss_tup_end, poss_comp_tup_beg, in_partition<number4>());
    thrust::stable_sort_by_key(_d_part_vec.begin(), _d_part_vec.end(), _d_poss_comp_ptr, part_sort());
    
    thrust::copy_if(vels_tup_beg, vels_tup_end, vels_comp_tup_beg, in_partition<number4>());
    thrust::stable_sort_by_key(_d_part_vec.begin(), _d_part_vec.end(), _d_vels_comp_ptr, part_sort());
    
    thrust::copy_if(Ls_tup_beg, Ls_tup_end, Ls_comp_tup_beg, in_partition<number4>());
    thrust::stable_sort_by_key(_d_part_vec.begin(), _d_part_vec.end(), _d_Ls_comp_ptr, part_sort());
    
    thrust::copy_if(orientations_tup_beg, orientations_tup_end, orientations_comp_tup_beg, in_partition<GPU_quat<number> >());
    thrust::stable_sort_by_key(_d_part_vec.begin(), _d_part_vec.end(), _d_orientations_comp_ptr, part_sort());
}

template<typename number, typename number4>
void MD_SP_CUDAMPIBackend<number, number4>::_check_partition()
{
    check_partition<number, number4>
        <<<this->_particles_kernel_cfg.blocks, this->_particles_kernel_cfg.threads_per_block>>>
        (this->_d_poss, _d_partition, this->_N, _part_center, _part_radius, this->_box_side, _boundary_size);
}

template<typename number, typename number4>
void MD_SP_CUDAMPIBackend<number, number4>::_count_partition()
{
    thrust::device_ptr<int> _d_partition_ptr(_d_partition);
    this->_P_N = thrust::count_if(_d_partition_ptr, _d_partition_ptr + this->_N, is_one());
    this->_B_N = thrust::count_if(_d_partition_ptr, _d_partition_ptr + this->_N, is_two());
}

template<typename number, typename number4>
void MD_SP_CUDAMPIBackend<number, number4>::_redistribute_particles()
{
    redistribute_particles<number, number4>
        <<<this->_particles_kernel_cfg.blocks, this->_particles_kernel_cfg.threads_per_block>>>
        (this->_d_poss, this->_d_vels, this->_d_Ls, this->_d_orientations,
         _d_buff_poss, _d_buff_vels, _d_buff_Ls, _d_buff_orientations, this->_N);
}

template<typename number, typename number4>
void MD_SP_CUDAMPIBackend<number, number4>::_exchange_particles()
{
    CUDA_SAFE_CALL( hipMemcpy(this->_h_comp_poss, this->_d_comp_poss, this->_N * sizeof(number4), hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy(this->_h_comp_vels, this->_d_comp_vels, this->_N * sizeof(number4), hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy(this->_h_comp_Ls, this->_d_comp_Ls, this->_N * sizeof(number4), hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy(this->_h_comp_orientations, this->_d_comp_orientations, this->_N * sizeof(GPU_quat<number>), hipMemcpyDeviceToHost) );
    
    int part_counts[_proc_size];
    int part_offsets[_proc_size];

    MPI_Allgather(&_P_N, 1, MPI_INT, &part_counts[0], 1, MPI_INT, MPI_COMM_WORLD);

    part_offsets[0] = 0;
    for (int i = 0; i < _proc_size; i++)
        part_offsets[i] = part_offsets[i - 1] + part_counts[i - 1];

    MPI_Allgatherv(_h_comp_poss, _P_N, this->_number4_type, _h_buff_poss,
                   part_counts, part_offsets, this->_number4_type, MPI_COMM_WORLD);
    MPI_Allgatherv(_h_comp_vels, _P_N, this->_number4_type, _h_buff_vels,
                   part_counts, part_offsets, this->_number4_type, MPI_COMM_WORLD);
    MPI_Allgatherv(_h_comp_Ls, _P_N, this->_number4_type, _h_buff_Ls,
                   part_counts, part_offsets, this->_number4_type, MPI_COMM_WORLD);
    MPI_Allgatherv(_h_comp_orientations, _P_N, this->_quat_type, _h_buff_orientations,
                   part_counts, part_offsets, this->_quat_type, MPI_COMM_WORLD);

    MPI_Barrier(MPI_COMM_WORLD);

    CUDA_SAFE_CALL( hipMemcpy(this->_d_buff_poss, this->_h_buff_poss, this->_N * sizeof(number4), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(this->_d_buff_vels, this->_h_buff_vels, this->_N * sizeof(number4), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(this->_d_buff_Ls, this->_h_buff_Ls, this->_N * sizeof(number4), hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(this->_d_buff_orientations, this->_h_buff_orientations, this->_N * sizeof(GPU_quat<number>), hipMemcpyHostToDevice) );
}

template<typename number, typename number4>
void MD_SP_CUDAMPIBackend<number, number4>::_compute_num_blocks(int N)
{
    this->_particles_kernel_cfg.blocks.x = N / this->_particles_kernel_cfg.threads_per_block + ((N % this->_particles_kernel_cfg.threads_per_block == 0) ? 0 : 1);
    this->_cuda_interaction->set_launch_cfg(this->_particles_kernel_cfg);
}

template<typename number, typename number4>
void MD_SP_CUDAMPIBackend<number, number4>::choose_device () {
	OX_LOG(Logger::LOG_INFO, "Choosing device automatically");

	int ndev = -1, trydev = 0;
	hipError_t ggg;
	hipDeviceProp_t tryprop;

	hipGetDeviceCount (&ndev);
	OX_LOG(Logger::LOG_INFO, "Computer has %i devices", ndev);

    int valid = 0;
    int devices[ndev];
    
	while (trydev < ndev) {
		tryprop = get_device_prop (trydev);
		OX_LOG(Logger::LOG_INFO, " -- device %i has properties %i.%i", trydev, tryprop.major, tryprop.minor);
		if (tryprop.major < 2 && tryprop.minor <= 2)
        {
			trydev++;
			continue;
		}
        else
        {
            devices[valid++] = trydev;
        }
        trydev++;
    }
    if (valid == 0) throw oxDNAException("No suitable devices available");
    OX_LOG(Logger::LOG_INFO, "Process %d has access to %d good GPU devices", this->_myid, valid);

    // if statement here to determine if it's one machine with several GPUs
    // or several machines each with a GPU

    trydev = this->_myid % valid;
    set_device (devices[trydev]);
    int * dummyptr;
    ggg = GpuUtils::LR_cudaMalloc<int> (& dummyptr, (size_t)sizeof(int));
    if (ggg == hipSuccess) {
        OX_LOG(Logger::LOG_INFO, "Process %d using device %i", this->_myid, trydev);
        hipFree (dummyptr);
    }
    else {
        throw oxDNAException("Unable to access device");
    }

	this->_device_prop = get_device_prop(trydev);
	this->_device_number = trydev;
}

template<typename number, typename number4>
void MD_SP_CUDAMPIBackend<number, number4>::_first_step() {
	first_step<number, number4>
		<<<this->_particles_kernel_cfg.blocks, this->_particles_kernel_cfg.threads_per_block>>>
		(this->_d_comp_poss, this->_d_comp_orientations, this->_d_list_poss, _d_comp_vels, _d_comp_Ls, this->_d_forces, this->_d_torques, this->_d_are_lists_old);
	CUT_CHECK_ERROR("_first_step error");
}

template<typename number, typename number4>
void MD_SP_CUDAMPIBackend<number, number4>::_forces_second_step() {
	_set_external_forces();
	this->_cuda_interaction->compute_forces(this->_cuda_lists, this->_d_comp_poss, this->_d_comp_orientations, this->_d_forces, this->_d_torques, this->_d_bonds);

	second_step<number, number4>
		<<<this->_particles_kernel_cfg.blocks, this->_particles_kernel_cfg.threads_per_block>>>
		(this->_d_comp_vels, this->_d_comp_Ls, this->_d_forces, this->_d_torques);
		CUT_CHECK_ERROR("second_step");
}

template<typename number, typename number4>
void MD_SP_CUDAMPIBackend<number, number4>::_set_external_forces() {
	set_external_forces<number, number4>
		<<<this->_particles_kernel_cfg.blocks, this->_particles_kernel_cfg.threads_per_block>>>
		(this->_d_comp_poss, this->_d_comp_orientations, this->_d_ext_forces, this->_d_forces, this->_d_torques, this->_curr_step, this->_max_ext_forces);
	CUT_CHECK_ERROR("set_external_forces");
}

template<typename number, typename number4>
void MD_SP_CUDAMPIBackend<number, number4>::_thermalize(llint curr_step) {
	//_cuda_thermostat->apply_cuda(this->_d_poss, this->_d_orientations, _d_vels, _d_Ls, curr_step); 
	//_cuda_thermostat->apply_cuda(this->_d_poss, this->_d_orientations, _d_vels, _d_Ls, this->_d_massinvs, curr_step); // TLF changed
}

template<typename number, typename number4>
void MD_SP_CUDAMPIBackend<number, number4>::sim_step(llint curr_step) {
	this->_curr_step = curr_step;
	get_time(&this->_timer, 0);

	get_time(&this->_timer, 2);
    this->_compute_num_blocks(this->_P_N);
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_N), &this->_P_N, sizeof(int)) );
	_first_step();
	hipDeviceSynchronize();
	get_time(&this->_timer, 3);

	get_time(&this->_timer, 4);
    this->_exchange_particles();
    
    this->_compute_num_blocks(this->_N);
    this->_redistribute_particles();
    hipDeviceSynchronize();

    this->_check_partition();
    hipDeviceSynchronize();

    this->_count_partition();
    this->_compactify_partition();
	get_time(&this->_timer, 5);

	get_time(&this->_timer, 6);
    this->_compute_num_blocks(this->_P_N + _B_N);
    int _PB_N = _P_N + _B_N;
    CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(verlet_N), &_PB_N, sizeof(int)) );
    
    this->_cuda_lists->update(this->_d_poss, this->_d_list_poss, this->_d_bonds);
    this->_d_are_lists_old[0] = false;
    this->_N_updates++;
    hipDeviceSynchronize();
	get_time(&this->_timer, 7);

	get_time(&this->_timer, 8);
	_forces_second_step();
	hipDeviceSynchronize();
	get_time(&this->_timer, 9);

	get_time(&this->_timer, 10);
    this->_compute_num_blocks(this->_P_N);
	_thermalize(curr_step);
	hipDeviceSynchronize();
	get_time(&this->_timer, 11);

	get_time(&this->_timer, 1);

	process_times(&this->_timer);
}

template<typename number, typename number4>
void MD_SP_CUDAMPIBackend<number, number4>::get_settings(input_file &inp) {
	MD_CUDABackend<number, number4>::get_settings(inp);

    this->_part_dims[0] = 1;
	getInputInt(&inp, "partition_dim_x", &_part_dims[0], 0);
	if (this->_part_dims[0] <= 0) throw oxDNAException("Invalid partition x dimension '%d'\n", _part_dims[0]);

	this->_part_dims[1] = 1;
	getInputInt(&inp, "partition_dim_y", &_part_dims[1], 0);
	if (this->_part_dims[1] <= 0) throw oxDNAException("Invalid partition y dimension '%d'\n", _part_dims[1]);

	this->_part_dims[2] = 1;
	getInputInt(&inp, "partition_dim_z", &_part_dims[2], 0);
	if (this->_part_dims[2] <= 0) throw oxDNAException("Invalid partition z dimension '%d'\n", _part_dims[2]);

    this->_boundary_size = 2.0f;
}

template<typename number, typename number4>
void MD_SP_CUDAMPIBackend<number, number4>::init(char conf_filename[256]){
	MD_CUDABackend<number, number4>::init(conf_filename);
    
    MPI_Comm_rank(MPI_COMM_WORLD, &_myid);
    MPI_Comm_size(MPI_COMM_WORLD, &_proc_size);

    this->choose_device();
    
    // initialize MPI types
    _init_mpi_type<number4>(_number4_type, _quat_type);

    _h_partition = new int[this->_N];
    _h_buff_poss = new number4[this->_N];
    _h_buff_vels = new number4[this->_N];
    _h_buff_Ls = new number4[this->_N];
    
    _h_comp_poss = new number4[this->_N];
    _h_comp_vels = new number4[this->_N];
    _h_comp_Ls = new number4[this->_N];
    
    _h_buff_orientations = new GPU_quat<number>[this->_N];
    _h_comp_orientations = new GPU_quat<number>[this->_N];
    
    
    CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<int>(&_d_partition, this->_N * sizeof(int)) );

    CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<number4>(&_d_comp_poss, this->_N * sizeof(float4)) );
    CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<number4>(&_d_comp_vels, this->_N * sizeof(float4)) );
    CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<number4>(&_d_comp_Ls, this->_N * sizeof(float4)) );

    CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<number4>(&_d_buff_poss, this->_N * sizeof(float4)) );
    CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<number4>(&_d_buff_vels, this->_N * sizeof(float4)) );
    CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<number4>(&_d_buff_Ls, this->_N * sizeof(float4)) );

    CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<GPU_quat<number> >(&_d_buff_orientations, this->_N * sizeof(GPU_quat<number>)) );
    CUDA_SAFE_CALL( GpuUtils::LR_cudaMalloc<GPU_quat<number> >(&_d_comp_orientations, this->_N * sizeof(GPU_quat<number>)) );

    if(this->_sort_every > 0) throw oxDNAException("CUDA + MPI and CUDA_sort_every > 0 are not compatible");

	this->_host_particles_to_gpu();
	this->_init_CUDA_MD_symbols();

	this->_cuda_thermostat->set_seed(lrand48());
	this->_cuda_thermostat->init(this->_N);

	OX_DEBUG("Allocated CUDA memory: %.2lf MBs", GpuUtils::get_allocated_mem_mb());

    // calculate partition coordinates
	this->_part_coords[0] = _myid									% _part_dims[0];
	this->_part_coords[1] = _myid / (_part_dims[0])					% _part_dims[1];
	this->_part_coords[2] = _myid / (_part_dims[0] * _part_dims[1]) % _part_dims[2];
    
    // Calculate partition bounds
	this->_part_size.x = this->_box_side / _part_dims[0];
	this->_part_size.y = this->_box_side / _part_dims[1];
	this->_part_size.z = this->_box_side / _part_dims[2];

	this->_part_origin.x = _part_size.x * _part_coords[0];
	this->_part_origin.y = _part_size.y * _part_coords[1];
	this->_part_origin.z = _part_size.z * _part_coords[2];

    this->_part_radius.x = _part_size.x / (number)2;
    this->_part_radius.y = _part_size.y / (number)2;
    this->_part_radius.z = _part_size.z / (number)2;
    
    this->_part_center.x = _part_origin.x + _part_radius.x;
    this->_part_center.y = _part_origin.y + _part_radius.y;
    this->_part_center.z = _part_origin.z + _part_radius.z;
    
    this->_compute_num_blocks(this->_N);
    this->_check_partition();
    hipDeviceSynchronize();

    this->_count_partition();
    this->_compactify_partition();
}

// template instantiations
template class MD_SP_CUDAMPIBackend<float, float4>;
template class MD_SP_CUDAMPIBackend<double, LR_double4>;
