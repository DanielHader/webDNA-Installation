#include "hip/hip_runtime.h"
/*
 * CUDANBLOCKInteraction.h
 *
 *  Created on: 26/may/2015
 *      Author: tyler
 */
#define PRINTV(s, v) printf("%s: %.2f, %.2f, %.2f \n", s, v.x, v.y, v.z);
#define PRINTVFULL(s, v) printf("%s: %f, %f, %f, %f\n", s, v.x, v.y, v.z, v.w);

#include "CUDANBLOCKInteraction.h"

//#include "CUDA_NBLOCK.cuh"
//#include "CUDA_DNA.cuh"
#include "../Lists/CUDASimpleVerletList.h"
#include "../Lists/CUDANoList.h"
#include "../../Interactions/DNA2Interaction.h"
#include "../cuda_utils/CUDA_lr_common.cuh"
#include <iostream>
#include <stdio.h>

__constant__ int MD_N[1];
__constant__ float MD_box_side[1];
__constant__ int MD_n_forces[1];

__constant__ float MD_hb_multi[1];
__constant__ float MD_F1_A[2];
__constant__ float MD_F1_RC[2];
__constant__ float MD_F1_R0[2];
__constant__ float MD_F1_BLOW[2];
__constant__ float MD_F1_BHIGH[2];
__constant__ float MD_F1_RLOW[2];
__constant__ float MD_F1_RHIGH[2];
__constant__ float MD_F1_RCLOW[2];
__constant__ float MD_F1_RCHIGH[2];
// 50 = 2 * 5 * 5
__constant__ float MD_F1_EPS[50];
__constant__ float MD_F1_SHIFT[50];

__constant__ float MD_F2_K[2];
__constant__ float MD_F2_RC[2];
__constant__ float MD_F2_R0[2];
__constant__ float MD_F2_BLOW[2];
__constant__ float MD_F2_RLOW[2];
__constant__ float MD_F2_RCLOW[2];
__constant__ float MD_F2_BHIGH[2];
__constant__ float MD_F2_RCHIGH[2];
__constant__ float MD_F2_RHIGH[2];

__constant__ float MD_F5_PHI_A[4];
__constant__ float MD_F5_PHI_B[4];
__constant__ float MD_F5_PHI_XC[4];
__constant__ float MD_F5_PHI_XS[4];

__constant__ float MD_dh_RC[1];
__constant__ float MD_dh_RHIGH[1];
__constant__ float MD_dh_prefactor[1];
__constant__ float MD_dh_B[1];
__constant__ float MD_dh_minus_kappa[1];
__constant__ bool MD_dh_half_charged_ends[1];


template<typename number, typename number4>
CUDANBLOCKInteraction<number, number4>::CUDANBLOCKInteraction() {

}

template<typename number, typename number4>
CUDANBLOCKInteraction<number, number4>::~CUDANBLOCKInteraction() {

}

template<typename number, typename number4>
void CUDANBLOCKInteraction<number, number4>::get_settings(input_file &inp) {
	_use_debye_huckel = false;
	_use_oxDNA2_coaxial_stacking = false;
	_use_oxDNA2_FENE = false;
	std::string inter_type;
	if (getInputString(&inp, "interaction_type", inter_type, 0) == KEY_FOUND){
		if (inter_type.compare("DNA2") == 0) {
			_use_debye_huckel = true;
			_use_oxDNA2_coaxial_stacking = true;
			_use_oxDNA2_FENE = true;
			// copy-pasted from the DNA2Interaction constructor
			this->_int_map[DEBYE_HUCKEL] = (number (DNAInteraction<number>::*)(BaseParticle<number> *p, BaseParticle<number> *q, LR_vector<number> *r, bool update_forces)) &DNA2Interaction<number>::_debye_huckel;
			// I assume these are needed. I think the interaction map is used for when the observables want to print energy
			this->_int_map[this->BACKBONE] = (number (DNAInteraction<number>::*)(BaseParticle<number> *p, BaseParticle<number> *q, LR_vector<number> *r, bool update_forces)) &DNA2Interaction<number>::_backbone;
			this->_int_map[this->COAXIAL_STACKING] = (number (DNAInteraction<number>::*)(BaseParticle<number> *p, BaseParticle<number> *q, LR_vector<number> *r, bool update_forces)) &DNA2Interaction<number>::_coaxial_stacking;

			// we don't need the F4_... terms as the macros are used in the CUDA_DNA.cuh file; this doesn't apply for the F2_K term
			this->F2_K[1] = CXST_K_OXDNA2;
			_debye_huckel_half_charged_ends = true;
			this->_grooving = true;
			// end copy from DNA2Interaction

			// copied from DNA2Interaction::get_settings() (CPU), the least bad way of doing things
			getInputNumber(&inp, "salt_concentration", &_salt_concentration, 1);
			getInputBool(&inp, "dh_half_charged_ends", &_debye_huckel_half_charged_ends, 0);
			
			// lambda-factor (the dh length at T = 300K, I = 1.0)
			_debye_huckel_lambdafactor = 0.3616455f;
			getInputFloat(&inp, "dh_lambda", &_debye_huckel_lambdafactor, 0);
			
			// the prefactor to the Debye-Huckel term
			_debye_huckel_prefactor = 0.0543f;
			getInputFloat(&inp, "dh_strength", &_debye_huckel_prefactor, 0);
			// End copy from DNA2Interaction
		}
	}

	// this needs to be here so that the default value of this->_grooving can be overwritten
	// NBLOCKInteraction<number>::get_settings(inp);
	DNAInteraction<number>::get_settings(inp);
}


struct TLF_TMP {
    bool grooving;
    bool use_debye_huckel;
    bool use_oxDNA2_coaxial_stacking;
    bool use_oxDNA2_FENE;
} tmp;
	
template<typename number, typename number4>
void CUDANBLOCKInteraction<number, number4>::cuda_init(number box_side, int N) {
	CUDABaseInteraction<number, number4>::cuda_init(box_side, N);
    //NBLOCKInteraction<number>::init();
	DNAInteraction<number>::init();


    // TLF STRUCT TO PASS VARS (future fields) into kernels, START
    tmp.grooving = true;
    tmp.use_debye_huckel = true;
    tmp.use_oxDNA2_coaxial_stacking = true;
    tmp.use_oxDNA2_FENE = true;
    // TLF STRUCT, END

	float f_copy = box_side;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_box_side), &f_copy, sizeof(float)) );
	f_copy = this->_hb_multiplier;
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_hb_multi), &f_copy, sizeof(float)) );

	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_N), &N, sizeof(int)) );

	number tmp[50];
	for(int i = 0; i < 2; i++) for(int j = 0; j < 5; j++) for(int k = 0; k < 5; k++) tmp[i*25 + j*5 + k] = this->F1_EPS[i][j][k];

	COPY_ARRAY_TO_CONSTANT(MD_F1_EPS, tmp, 50);

	for(int i = 0; i < 2; i++) for(int j = 0; j < 5; j++) for(int k = 0; k < 5; k++) tmp[i*25 + j*5 + k] = this->F1_SHIFT[i][j][k];

	COPY_ARRAY_TO_CONSTANT(MD_F1_SHIFT, tmp, 50);

	COPY_ARRAY_TO_CONSTANT(MD_F1_A, this->F1_A, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_RC, this->F1_RC, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_R0, this->F1_R0, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_BLOW, this->F1_BLOW, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_BHIGH, this->F1_BHIGH, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_RLOW, this->F1_RLOW, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_RHIGH, this->F1_RHIGH, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_RCLOW, this->F1_RCLOW, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F1_RCHIGH, this->F1_RCHIGH, 2);

	COPY_ARRAY_TO_CONSTANT(MD_F2_K, this->F2_K, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_RC, this->F2_RC, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_R0, this->F2_R0, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_BLOW, this->F2_BLOW, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_BHIGH, this->F2_BHIGH, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_RLOW, this->F2_RLOW, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_RHIGH, this->F2_RHIGH, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_RCLOW, this->F2_RCLOW, 2);
	COPY_ARRAY_TO_CONSTANT(MD_F2_RCHIGH, this->F2_RCHIGH, 2);

	COPY_ARRAY_TO_CONSTANT(MD_F5_PHI_A, this->F5_PHI_A, 4);
	COPY_ARRAY_TO_CONSTANT(MD_F5_PHI_B, this->F5_PHI_B, 4);
	COPY_ARRAY_TO_CONSTANT(MD_F5_PHI_XC, this->F5_PHI_XC, 4);
	COPY_ARRAY_TO_CONSTANT(MD_F5_PHI_XS, this->F5_PHI_XS, 4);

	if(this->_use_edge) CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_n_forces), &this->_n_forces, sizeof(int)) );
	if (_use_debye_huckel){
		// copied from DNA2Interaction::init() (CPU), the least bad way of doing things
		// We wish to normalise with respect to T=300K, I=1M. 300K=0.1 s.u. so divide this->_T by 0.1
		number lambda = _debye_huckel_lambdafactor * sqrt(this->_T / 0.1f) / sqrt(_salt_concentration);
		// RHIGH gives the distance at which the smoothing begins
		_debye_huckel_RHIGH = 3.0 * lambda;
		_minus_kappa = -1.0/lambda;

		// these are just for convenience for the smoothing parameter computation
		number x = _debye_huckel_RHIGH;
		number q = _debye_huckel_prefactor;
		number l = lambda;

		// compute the some smoothing parameters
		_debye_huckel_B = -(exp(-x/l) * q * q * (x + l)*(x+l) )/(-4.*x*x*x * l * l * q );
		_debye_huckel_RC = x*(q*x + 3. * q* l )/(q * (x+l));

		number debyecut;
		if (this->_grooving){
			debyecut = 2.0f * sqrt((POS_MM_BACK1)*(POS_MM_BACK1) + (POS_MM_BACK2)*(POS_MM_BACK2)) + _debye_huckel_RC;
		}
		else{
			debyecut =  2.0f * sqrt(SQR(POS_BACK)) + _debye_huckel_RC;
		}
		// the cutoff radius for the potential should be the larger of rcut and debyecut
		if (debyecut > this->_rcut){
			this->_rcut = debyecut;
			this->_sqr_rcut = debyecut*debyecut;
		}
		// End copy from DNA2Interaction

		CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_RC), &_debye_huckel_RC, sizeof(float)) );
		CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_RHIGH), &_debye_huckel_RHIGH, sizeof(float)) );
		CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_prefactor), &_debye_huckel_prefactor, sizeof(float)) );
		CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_B), &_debye_huckel_B, sizeof(float)) );
		CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_minus_kappa), &_minus_kappa, sizeof(float)) );
		CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(MD_dh_half_charged_ends), &_debye_huckel_half_charged_ends, sizeof(bool)) );
	}
}


/* System constants */

template<typename number, typename number4>
__forceinline__ __device__ void _excluded_volume(const number4 &r, number4 &F, number sigma, number rstar, number b, number rc) {
	number rsqr = CUDA_DOT(r, r);

	F.x = F.y = F.z = F.w = (number) 0.f;
	if(rsqr < SQR(rc)) {
		if(rsqr > SQR(rstar)) {
			number rmod = sqrt(rsqr);
			number rrc = rmod - rc;
			number fmod = 2.f * EXCL_EPS * b * rrc / rmod;
			F.x = r.x * fmod;
			F.y = r.y * fmod;
			F.z = r.z * fmod;
			F.w = EXCL_EPS * b * SQR(rrc);
		}
		else {
			number lj_part = CUB(SQR(sigma)/rsqr);
			number fmod = 24.f * EXCL_EPS * (lj_part - 2.f*SQR(lj_part)) / rsqr;
			F.x = r.x * fmod;
			F.y = r.y * fmod;
			F.z = r.z * fmod;
			F.w = 4.f * EXCL_EPS * (SQR(lj_part) - lj_part);
		}
	}
}

template<typename number>
__forceinline__ __device__ number _f1(number r, int type, int n3, int n5) {
	number val = (number) 0.f;
	if(r < MD_F1_RCHIGH[type]) {
		int eps_index = 25 * type + n3 * 5 + n5;
		if(r > MD_F1_RHIGH[type]) {
			val = MD_F1_EPS[eps_index] * MD_F1_BHIGH[type] * SQR(r - MD_F1_RCHIGH[type]);
		}
		else if(r > MD_F1_RLOW[type]) {
			number tmp = 1.f - expf(-(r - MD_F1_R0[type]) * MD_F1_A[type]);
			val = MD_F1_EPS[eps_index] * SQR(tmp) - MD_F1_SHIFT[eps_index];
		}
		else if(r > MD_F1_RCLOW[type]) {
			val = MD_F1_EPS[eps_index] * MD_F1_BLOW[type] * SQR(r - MD_F1_RCLOW[type]);
		}
	}

	return val;
}

template<typename number>
__forceinline__ __device__ number _f1D(number r, int type, int n3, int n5) {
	number val = (number) 0.f;
	int eps_index = 0;
	if(r < MD_F1_RCHIGH[type]) {
		eps_index = 25 * type + n3 * 5 + n5;
		if(r > MD_F1_RHIGH[type]) {
			val = 2.f * MD_F1_BHIGH[type] * (r - MD_F1_RCHIGH[type]);
		}
		else if(r > MD_F1_RLOW[type]) {
			number tmp = expf(-(r - MD_F1_R0[type]) * MD_F1_A[type]);
			val = 2.f * (1.f - tmp) * tmp * MD_F1_A[type];
		}
		else if(r > MD_F1_RCLOW[type]) {
			val = 2.f * MD_F1_BLOW[type] * (r - MD_F1_RCLOW[type]);
		}
	}

	return MD_F1_EPS[eps_index] * val;
}

template<typename number>
__forceinline__ __device__ number _f2(number r, int type) {
    number val = (number) 0.f;
    if (r < MD_F2_RCHIGH[type]) {
	    if (r > MD_F2_RHIGH[type]) {
		    val = MD_F2_K[type] * MD_F2_BHIGH[type] * SQR(r - MD_F2_RCHIGH[type]);
	    }
	    else if (r > MD_F2_RLOW[type]) {
		    val = (MD_F2_K[type] * 0.5f) * (SQR(r - MD_F2_R0[type]) - SQR(MD_F2_RC[type] - MD_F2_R0[type]));
	    }
	    else if (r > MD_F2_RCLOW[type]) {
		    val = MD_F2_K[type] * MD_F2_BLOW[type] * SQR(r - MD_F2_RCLOW[type]);
	    }
    }
    return val;
}

template<typename number>
__forceinline__ __device__ number _f2D(number r, int type) {
    number val = (number) 0.f;
    if (r < MD_F2_RCHIGH[type]) {
	    if (r > MD_F2_RHIGH[type]) {
		    val = 2.f * MD_F2_K[type] * MD_F2_BHIGH[type] * (r - MD_F2_RCHIGH[type]);
	    }
	    else if (r > MD_F2_RLOW[type]) {
		    val = MD_F2_K[type] * (r - MD_F2_R0[type]);
	    }
	    else if (r > MD_F2_RCLOW[type]) {
		    val = 2.f * MD_F2_K[type] * MD_F2_BLOW[type] * (r - MD_F2_RCLOW[type]);
	    }
    }
    return val;
}

template<typename number>
__forceinline__ __device__ number _f4(number t, float t0, float ts, float tc, float a, float b) {
	number val = (number) 0.f;
	t -= t0;
	if(t < 0) t = -t;

	if(t < tc) {
		if(t > ts) {
			// smoothing
			val = b * SQR(tc - t);
		}
		else val = (number) 1.f - a * SQR(t);
	}

	return val;
}

template<typename number>
__forceinline__ __device__ number _f4_pure_harmonic(number t, float a, float b) {
	// for getting a f4t1 function with a continuous derivative that is less disruptive to the potential
	number val = (number) 0.f;
	t -= b;
	if(t < 0) val = (number) 0.f;
	else val = (number) a * SQR(t);

	return val;
}

template<typename number>
__forceinline__ __device__ number _f4Dsin(number t, float t0, float ts, float tc, float a, float b) {
	number val = (number) 0.f;
	number tt0 = t - t0;
	// this function is a parabola centered in t0. If tt0 < 0 then the value of the function
	// is the same but the value of its derivative has the opposite sign, so m = -1
	number m = copysignf((number)1.f, tt0);
	tt0 = copysignf(tt0, (number)1.f);

	if(tt0 < tc) {
		number sint = sinf(t);
		if(tt0 > ts) {
			// smoothing
			val = b * (tt0 - tc) / sint;
		}
		else {
			if(SQR(sint) > 1e-12f) val = -a * tt0 / sint;
			else val = -a;
		}
	}

	return 2.f * m * val;
}

template<typename number>
__forceinline__ __device__ number _f4Dsin_pure_harmonic(number t, float a, float b) {
	// for getting a f4t1 function with a continuous derivative that is less disruptive to the potential
	number val = (number) 0.f;
	number tt0 = t - b;
	if(tt0 < 0) val = (number) 0.f;
	else {
		number sint = sin(t);
		if (SQR(sint) > 1e-12) val = (number) 2 * a * tt0 / sint;
		else val = (number) 2 * a;
	}
	
	return val;
}

template<typename number>
__forceinline__ __device__ number _f5(number f, int type) {
	number val = (number) 0.f;

	if(f > MD_F5_PHI_XC[type]) {
		if(f < MD_F5_PHI_XS[type]) {
			val = MD_F5_PHI_B[type] * SQR(MD_F5_PHI_XC[type] - f);
		}
		else if(f < 0.f) {
			val = (number) 1.f - MD_F5_PHI_A[type] * SQR(f);
		}
		else val = 1.f;
	}

	return val;
}

template<typename number>
__forceinline__ __device__ number _f5D(number f, int type) {
	number val = (number) 0.f;

	if(f > MD_F5_PHI_XC[type]) {
		if(f < MD_F5_PHI_XS[type]) {
			val = 2.f * MD_F5_PHI_B[type] * (f - MD_F5_PHI_XC[type]);
		}
		else if(f < 0.f) {
			val = (number) -2.f * MD_F5_PHI_A[type] * f;
		}
	}

	return val;
}

template <typename number, typename number4>
__device__ number4 minimum_image(const number4 &r_i, const number4 &r_j) {
	number dx = r_j.x - r_i.x;
	number dy = r_j.y - r_i.y;
	number dz = r_j.z - r_i.z;

	dx -= floorf(dx/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];
	dy -= floorf(dy/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];
	dz -= floorf(dz/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];

	return make_number4<number, number4>(dx, dy, dz, (number) 0.f);
}

template <typename number, typename number4>
__device__ number quad_minimum_image_dist(const number4 &r_i, const number4 &r_j) {
	number dx = r_j.x - r_i.x;
	number dy = r_j.y - r_i.y;
	number dz = r_j.z - r_i.z;

	dx -= floorf(dx/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];
	dy -= floorf(dy/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];
	dz -= floorf(dz/MD_box_side[0] + (number) 0.5f) * MD_box_side[0];

	return dx*dx + dy*dy + dz*dz;
}

template <typename number, typename number4, bool qIsN3>
__device__ void _bonded_excluded_volume(number4 &r, number4 &n3pos_base, number4 &n3pos_back, number4 &n5pos_base, number4 &n5pos_back, number4 &F, number4 &T) {
	number4 Ftmp;
	// BASE-BASE
	number4 rcenter = r + n3pos_base - n5pos_base;
	_excluded_volume(rcenter, Ftmp, EXCL_S2, EXCL_R2, EXCL_B2, EXCL_RC2);
	number4 torquep1 = (qIsN3) ? _cross<number, number4>(n5pos_base, Ftmp) : _cross<number, number4>(n3pos_base, Ftmp);
	F += Ftmp;

	// n5-BASE vs. n3-BACK
	rcenter = r + n3pos_back - n5pos_base;
	_excluded_volume(rcenter, Ftmp, EXCL_S3, EXCL_R3, EXCL_B3, EXCL_RC3);
	number4 torquep2 = (qIsN3) ? _cross<number, number4>(n5pos_base, Ftmp) : _cross<number, number4>(n3pos_back, Ftmp);
	F += Ftmp;

	// n5-BACK vs. n3-BASE
	rcenter = r + n3pos_base - n5pos_back;
	_excluded_volume(rcenter, Ftmp, EXCL_S4, EXCL_R4, EXCL_B4, EXCL_RC4);
	number4 torquep3 = (qIsN3) ? _cross<number, number4>(n5pos_back, Ftmp) : _cross<number, number4>(n3pos_base, Ftmp);
	F += Ftmp;

	T += torquep1 + torquep2 + torquep3;
}
 
template <typename number, typename number4>
__device__ void _np_nucleotide_bonded(const number4 &n5pos, number4 &n5x, number4 &n5y, number4 &n5z, const number4 &n3pos, number4 &n3x, number4 &n3y, number4 &n3z, number4 &F, number4 &T, number4 &np_F, number4 &np_T) {

    // TLF quick notes on the fuction:
    // n5 / p          is the nucleotide
    // n3 / q          is the nano particle
    // grooving        is assumed true
    // use_oxDNA2_FENE is assumed true

    const number np_radius = 2.440716;
    const number stiffness = 8; 

    // Get binding location of strand to np, assuming single strand ATM.
    const number4 w = n3x * np_radius;

    // Translate NP pos to edge of sphere
    const number4 v = n3pos + w;

    // Pos of nucl backbone, grooving == true
	const number4 n5pos_back = n5x * POS_MM_BACK1 + n5y * POS_MM_BACK2;
	const number4 u          = n5pos + n5pos_back;

    // Get distance between the backbone and the pt. where it is bound
    const number4 dr       = u - v;
    const number4 force    = dr * stiffness;
    const number4 np_force = force * (number).002370; // massinv

    /*
     * Shouldn't work but swapping np_force for force
     * in the calculation of np_torque, but applying np_force to np_F 
     * allows the simulation to maintain moderate energy levels for longer.
     */
    const number4 np_torque = _cross<number, number4>(w, np_force);
    //const number4 np_torque   = _cross<number, number4>(w, force);
    const number4 nucl_torque = _cross<number, number4>(n5pos_back, force);
    const number module = _module<number, number4>(dr);
    //const number energy = SQR(module) * ((number) 0.5) * stiffness;
    const number energy = SQR(module) * stiffness;

    F      -= force;
    np_F   += np_force;

    //T      -= _vectors_transpose_number4_product(n5x, n5y, n5z, nucl_torque);
    T      -= nucl_torque;
    np_T   += _vectors_transpose_number4_product(n3x, n3y, n3z, np_torque);

    F.w    = energy;
    np_F.w = energy;
}

template <typename number, typename number4, bool qIsN3>
__device__ void _bonded_part(const number4 &n5pos, number4 &n5x, number4 &n5y, number4 &n5z, const number4 &n3pos, number4 &n3x, number4 &n3y, number4 &n3z, number4 &F, number4 &T, bool grooving, bool use_oxDNA2_FENE) {

	int n3type = get_particle_type<number, number4>(n3pos);
	int n5type = get_particle_type<number, number4>(n5pos);

	number4 r = make_number4<number, number4>(n3pos.x - n5pos.x, n3pos.y - n5pos.y, n3pos.z - n5pos.z, (number) 0);

    // grooving is assumed true TLF
	number4 n5pos_back = n5x * POS_MM_BACK1 + n5y * POS_MM_BACK2;

    /*
	number4 n5pos_back;
	if(grooving) n5pos_back = n5x * POS_MM_BACK1 + n5y * POS_MM_BACK2;
	else n5pos_back = n5x * POS_BACK;
    */

	number4 n5pos_base = n5x * POS_BASE;
	number4 n5pos_stack = n5x * POS_STACK;

    // grooving is assumed true TLF
	number4 n3pos_back = n3x * POS_MM_BACK1 + n3y * POS_MM_BACK2;

    /*
	number4 n3pos_back;
	if(grooving) n3pos_back = n3x * POS_MM_BACK1 + n3y * POS_MM_BACK2;
	else n3pos_back = n3x * POS_BACK;
    */
	number4 n3pos_base = n3x * POS_BASE;
	number4 n3pos_stack = n3x * POS_STACK;

	number4 rback = r + n3pos_back - n5pos_back;
	number rbackmod = _module<number, number4>(rback);

    // use_oxDNA2_FENE is assumed true TLF
	number rbackr0 = rbackmod - FENE_R0_OXDNA2;

    /*
	number rbackr0;
	if (use_oxDNA2_FENE) rbackr0 = rbackmod - FENE_R0_OXDNA2;
	else rbackr0 = rbackmod - FENE_R0_OXDNA;
    */

	number4 Ftmp = rback * ((FENE_EPS * rbackr0  / (FENE_DELTA2 - SQR(rbackr0))) / rbackmod);
	Ftmp.w = -FENE_EPS * ((number)0.5f) * logf(1 - SQR(rbackr0) / FENE_DELTA2);

	number4 Ttmp = (qIsN3) ? _cross<number, number4>(n5pos_back, Ftmp) : _cross<number, number4>(n3pos_back, Ftmp);
	// EXCLUDED VOLUME
	_bonded_excluded_volume<number, number4, qIsN3>(r, n3pos_base, n3pos_back, n5pos_base, n5pos_back, Ftmp, Ttmp);

	if(qIsN3) {
		F += Ftmp;
		T += Ttmp;
	}
	else {
		F -= Ftmp;
		T -= Ttmp;
	}

	// STACKING
	number4 rstack = r + n3pos_stack - n5pos_stack;
	number rstackmod = _module<number, number4>(rstack);
	number4 rstackdir = make_number4<number, number4>(rstack.x / rstackmod, rstack.y / rstackmod, rstack.z / rstackmod, 0);
	// This is the position the backbone would have with major-minor grooves the same width.
	// We need to do this to implement different major-minor groove widths because rback is
	// used as a reference point for things that have nothing to do with the actual backbone
	// position (in this case, the stacking interaction).
	number4 rbackref = r + n3x * POS_BACK - n5x * POS_BACK;
	number rbackrefmod = _module<number, number4>(rbackref);

	number t4 = CUDA_LRACOS(CUDA_DOT(n3z, n5z));
	number t5 = CUDA_LRACOS(CUDA_DOT(n5z, rstackdir));
	number t6 = CUDA_LRACOS(-CUDA_DOT(n3z, rstackdir));
	number cosphi1 = CUDA_DOT(n5y, rbackref) / rbackrefmod;
	number cosphi2 = CUDA_DOT(n3y, rbackref) / rbackrefmod;

	// functions
	number f1 = _f1(rstackmod, STCK_F1, n3type, n5type);
	number f4t4 = _f4(t4, STCK_THETA4_T0, STCK_THETA4_TS, STCK_THETA4_TC, STCK_THETA4_A, STCK_THETA4_B);
	number f4t5 = _f4(PI - t5, STCK_THETA5_T0, STCK_THETA5_TS, STCK_THETA5_TC, STCK_THETA5_A, STCK_THETA5_B);
	number f4t6 = _f4(t6, STCK_THETA6_T0, STCK_THETA6_TS, STCK_THETA6_TC, STCK_THETA6_A, STCK_THETA6_B);
	number f5phi1 = _f5(cosphi1, STCK_F5_PHI1);
	number f5phi2 = _f5(cosphi2, STCK_F5_PHI2);

	number energy = f1 * f4t4 * f4t5 * f4t6 * f5phi1 * f5phi2;

	if(energy != (number) 0) {
		// and their derivatives
		number f1D = _f1D(rstackmod, STCK_F1, n3type, n5type);
		number f4t4Dsin = _f4Dsin(t4, STCK_THETA4_T0, STCK_THETA4_TS, STCK_THETA4_TC, STCK_THETA4_A, STCK_THETA4_B);
			number f4t5Dsin = _f4Dsin(PI - t5, STCK_THETA5_T0, STCK_THETA5_TS, STCK_THETA5_TC, STCK_THETA5_A, STCK_THETA5_B);
			number f4t6Dsin = _f4Dsin(t6, STCK_THETA6_T0, STCK_THETA6_TS, STCK_THETA6_TC, STCK_THETA6_A, STCK_THETA6_B);
		number f5phi1D = _f5D(cosphi1, STCK_F5_PHI1);
		number f5phi2D = _f5D(cosphi2, STCK_F5_PHI2);

		// RADIAL
		Ftmp = rstackdir * (energy * f1D / f1);

		// THETA 5
		Ftmp += (n5z - cosf(t5) * rstackdir) * (energy * f4t5Dsin / (f4t5 * rstackmod));

		// THETA 6
		Ftmp += (n3z + cosf(t6) * rstackdir) * (energy * f4t6Dsin / (f4t6 * rstackmod));

		// COS PHI 1
		// here particle p is referred to using the a while particle q is referred with the b
		number ra2 = CUDA_DOT(rstackdir, n5y);
		number ra1 = CUDA_DOT(rstackdir, n5x);
		number rb1 = CUDA_DOT(rstackdir, n3x);
		number a2b1 = CUDA_DOT(n5y, n3x);
		number dcosphi1dr = (SQR(rstackmod)*ra2 - ra2*SQR(rbackrefmod) - rstackmod*(a2b1 + ra2*(-ra1 + rb1))*GAMMA + a2b1*(-ra1 + rb1)*SQR(GAMMA))/(SQR(rbackrefmod)*rbackrefmod);
		number dcosphi1dra1 = rstackmod*GAMMA*(rstackmod*ra2 - a2b1*GAMMA)/(SQR(rbackrefmod)*rbackrefmod);
		number dcosphi1dra2 = -rstackmod / rbackrefmod;
		number dcosphi1drb1 = -(rstackmod*GAMMA*(rstackmod*ra2 - a2b1*GAMMA))/(SQR(rbackrefmod)*rbackrefmod);
		number dcosphi1da1b1 = SQR(GAMMA)*(-rstackmod*ra2 + a2b1*GAMMA)/(SQR(rbackrefmod)*rbackrefmod);
		number dcosphi1da2b1 = GAMMA / rbackrefmod;

		number force_part_phi1 = energy * f5phi1D / f5phi1;

		Ftmp -= (rstackdir * dcosphi1dr +
			    ((n5y - ra2*rstackdir) * dcosphi1dra2 +
				(n5x - ra1*rstackdir) * dcosphi1dra1 +
				(n3x - rb1*rstackdir) * dcosphi1drb1) / rstackmod) * force_part_phi1;

		// COS PHI 2
		// here particle p -> b, particle q -> a
		ra2 = CUDA_DOT(rstackdir, n3y);
		ra1 = rb1;
		rb1 = CUDA_DOT(rstackdir, n5x);
		a2b1 = CUDA_DOT(n3y, n5x);
		number dcosphi2dr = ((rstackmod*ra2 + a2b1*GAMMA)*(rstackmod + (rb1 - ra1)*GAMMA) - ra2*SQR(rbackrefmod))/(SQR(rbackrefmod)*rbackrefmod);
		number dcosphi2dra1 = -rstackmod*GAMMA*(rstackmod*ra2 + a2b1*GAMMA)/(SQR(rbackrefmod)*rbackrefmod);
		number dcosphi2dra2 = -rstackmod / rbackrefmod;
		number dcosphi2drb1 = (rstackmod*GAMMA*(rstackmod*ra2 + a2b1*GAMMA))/(SQR(rbackrefmod)*rbackrefmod);
		number dcosphi2da1b1 = -SQR(GAMMA)*(rstackmod*ra2 + a2b1*GAMMA)/(SQR(rbackrefmod)*rbackrefmod);
		number dcosphi2da2b1 = -GAMMA / rbackrefmod;

		number force_part_phi2 = energy * f5phi2D / f5phi2;

		Ftmp -= (rstackdir * dcosphi2dr +
			    ((n3y - rstackdir * ra2) * dcosphi2dra2 +
				(n3x - rstackdir * ra1) * dcosphi2dra1 +
				(n5x - rstackdir * rb1) * dcosphi2drb1) / rstackmod) * force_part_phi2;

		if(qIsN3) Ttmp = _cross<number, number4>(n5pos_stack, Ftmp);
		else Ttmp = _cross<number, number4>(n3pos_stack, Ftmp);

		// THETA 4
		Ttmp += _cross<number, number4>(n3z, n5z) * (-energy * f4t4Dsin / f4t4);

		// PHI 1 & PHI 2
		if(qIsN3) {
			Ttmp += (-force_part_phi1 * dcosphi1dra2) * _cross<number, number4>(rstackdir, n5y)
				-_cross<number, number4>(rstackdir, n5x) * force_part_phi1 * dcosphi1dra1;

			Ttmp += (-force_part_phi2 * dcosphi2drb1) * _cross<number, number4>(rstackdir, n5x);
		}
		else {
			Ttmp += force_part_phi1 * dcosphi1drb1 * _cross<number, number4>(rstackdir, n3x);

			Ttmp += force_part_phi2 * dcosphi2dra2 * _cross<number, number4>(rstackdir, n3y) +
				force_part_phi2 * dcosphi2dra1 * _cross<number, number4>(rstackdir, n3x);
		}

		Ttmp += force_part_phi1 * dcosphi1da2b1 * _cross<number, number4>(n5y, n3x)
			+ _cross<number, number4>(n5x, n3x) * force_part_phi1 * dcosphi1da1b1;

		Ttmp += force_part_phi2 * dcosphi2da2b1 * _cross<number, number4>(n5x, n3y) +
			_cross<number, number4>(n5x, n3x) * force_part_phi2 * dcosphi2da1b1;

		Ftmp.w = energy;
		if(qIsN3) {
			// THETA 5
			Ttmp += _cross<number, number4>(rstackdir, n5z) * energy * f4t5Dsin / f4t5;

			T += Ttmp;
			F += Ftmp;
		}
		else {
			// THETA 6
			Ttmp += _cross<number, number4>(rstackdir, n3z) * (-energy * f4t6Dsin / f4t6);

			T -= Ttmp;
			F -= Ftmp;
		}
	}
}

template <typename number, typename number4>
__device__ void _particle_particle_interaction(number4 ppos, number4 a1, number4 a2, number4 a3, number4 qpos, number4 b1, number4 b2, number4 b3, number4 &F, number4 &T, bool grooving, bool use_debye_huckel, bool use_oxDNA2_coaxial_stacking, LR_bonds pbonds, LR_bonds qbonds, int pind, int qind) {
	int ptype = get_particle_type<number, number4>(ppos);
	int qtype = get_particle_type<number, number4>(qpos);
	int pbtype = get_particle_btype<number, number4>(ppos);
	int qbtype = get_particle_btype<number, number4>(qpos);
	int int_type = pbtype + qbtype;

	number4 r = minimum_image<number, number4>(ppos, qpos);

	number4 ppos_back;
	if(grooving) ppos_back = POS_MM_BACK1 * a1 + POS_MM_BACK2 * a2;
	else ppos_back = POS_BACK * a1;
	number4 ppos_base = POS_BASE * a1;
	number4 ppos_stack = POS_STACK * a1;

	number4 qpos_back;
	if(grooving) qpos_back = POS_MM_BACK1 * b1 + POS_MM_BACK2 * b2;
	else qpos_back = POS_BACK * b1;
	number4 qpos_base = POS_BASE * b1;
	number4 qpos_stack = POS_STACK * b1;

	number old_Tw = T.w;

	// excluded volume
	// BACK-BACK
	number4 Ftmp = make_number4<number, number4>(0, 0, 0, 0);
	number4 rbackbone = r + qpos_back - ppos_back;
	_excluded_volume(rbackbone, Ftmp, EXCL_S1, EXCL_R1, EXCL_B1, EXCL_RC1);
	number4 Ttmp = _cross<number, number4>(ppos_back, Ftmp);
	_bonded_excluded_volume<number, number4, true>(r, qpos_base, qpos_back, ppos_base, ppos_back, Ftmp, Ttmp);

	F += Ftmp;

	// HYDROGEN BONDING
	number hb_energy = (number) 0;
	number4 rhydro = r + qpos_base - ppos_base;
	number rhydromodsqr = CUDA_DOT(rhydro, rhydro);
	if(int_type == 3 && SQR(HYDR_RCLOW) < rhydromodsqr && rhydromodsqr < SQR(HYDR_RCHIGH)) {
		number hb_multi = (abs(qbtype) >= 300 && abs(pbtype) >= 300) ? MD_hb_multi[0] : 1.f;
		// versor and magnitude of the base-base separation
	  	number rhydromod = sqrtf(rhydromodsqr);
	  	number4 rhydrodir = rhydro / rhydromod;

		// angles involved in the HB interaction
		number t1 = CUDA_LRACOS(-CUDA_DOT(a1, b1));
		number t2 = CUDA_LRACOS(-CUDA_DOT(b1, rhydrodir));
		number t3 = CUDA_LRACOS(CUDA_DOT(a1, rhydrodir));
		number t4 = CUDA_LRACOS(CUDA_DOT(a3, b3));
		number t7 = CUDA_LRACOS(-CUDA_DOT(rhydrodir, b3));
		number t8 = CUDA_LRACOS(CUDA_DOT(rhydrodir, a3));

	 	 // functions called at their relevant arguments
		number f1 = hb_multi * _f1(rhydromod, HYDR_F1, ptype, qtype);
		number f4t1 = _f4(t1, HYDR_THETA1_T0, HYDR_THETA1_TS, HYDR_THETA1_TC, HYDR_THETA1_A, HYDR_THETA1_B);
		number f4t2 = _f4(t2, HYDR_THETA2_T0, HYDR_THETA2_TS, HYDR_THETA2_TC, HYDR_THETA2_A, HYDR_THETA2_B);
		number f4t3 = _f4(t3, HYDR_THETA3_T0, HYDR_THETA3_TS, HYDR_THETA3_TC, HYDR_THETA3_A, HYDR_THETA3_B);
		number f4t4 = _f4(t4, HYDR_THETA4_T0, HYDR_THETA4_TS, HYDR_THETA4_TC, HYDR_THETA4_A, HYDR_THETA4_B);
		number f4t7 = _f4(t7, HYDR_THETA7_T0, HYDR_THETA7_TS, HYDR_THETA7_TC, HYDR_THETA7_A, HYDR_THETA7_B);
		number f4t8 = _f4(t8, HYDR_THETA8_T0, HYDR_THETA8_TS, HYDR_THETA8_TC, HYDR_THETA8_A, HYDR_THETA8_B);

		hb_energy = f1 * f4t1 * f4t2 * f4t3 * f4t4 * f4t7 * f4t8;

		if(hb_energy < (number) 0) {
			// derivatives called at the relevant arguments
			number f1D = hb_multi * _f1D(rhydromod, HYDR_F1, ptype, qtype);
			number f4t1Dsin = - _f4Dsin(t1, HYDR_THETA1_T0, HYDR_THETA1_TS, HYDR_THETA1_TC, HYDR_THETA1_A, HYDR_THETA1_B);
			number f4t2Dsin = - _f4Dsin(t2, HYDR_THETA2_T0, HYDR_THETA2_TS, HYDR_THETA2_TC, HYDR_THETA2_A, HYDR_THETA2_B);
			number f4t3Dsin = _f4Dsin(t3, HYDR_THETA3_T0, HYDR_THETA3_TS, HYDR_THETA3_TC, HYDR_THETA3_A, HYDR_THETA3_B);
			number f4t4Dsin = _f4Dsin(t4, HYDR_THETA4_T0, HYDR_THETA4_TS, HYDR_THETA4_TC, HYDR_THETA4_A, HYDR_THETA4_B);
			number f4t7Dsin = - _f4Dsin(t7, HYDR_THETA7_T0, HYDR_THETA7_TS, HYDR_THETA7_TC, HYDR_THETA7_A, HYDR_THETA7_B);
			number f4t8Dsin = _f4Dsin(t8, HYDR_THETA8_T0, HYDR_THETA8_TS, HYDR_THETA8_TC, HYDR_THETA8_A, HYDR_THETA8_B);

			// RADIAL PART
			Ftmp = rhydrodir * hb_energy * f1D / f1;

			// TETA4; t4 = LRACOS (a3 * b3);
			Ttmp -= _cross<number, number4>(a3, b3) * (-hb_energy * f4t4Dsin / f4t4);

			// TETA1; t1 = LRACOS (-a1 * b1);
			Ttmp -= _cross<number, number4>(a1, b1) * (- hb_energy * f4t1Dsin / f4t1);

			// TETA2; t2 = LRACOS (-b1 * rhydrodir);
			Ftmp -= (b1 + rhydrodir * cosf(t2)) * (hb_energy * f4t2Dsin / (f4t2 * rhydromod));

			// TETA3; t3 = LRACOS (a1 * rhydrodir);
			number part = - hb_energy * f4t3Dsin / f4t3;
			Ftmp -= (a1 - rhydrodir * cosf(t3)) * (-part / rhydromod);
			Ttmp += _cross<number, number4>(rhydrodir, a1) * part;

			// THETA7; t7 = LRACOS (-rhydrodir * b3);
			Ftmp -= (b3 + rhydrodir * cosf(t7)) * (hb_energy * f4t7Dsin / (f4t7 * rhydromod));

			// THETA 8; t8 = LRACOS (rhydrodir * a3);
			part = - hb_energy * f4t8Dsin / f4t8;
			Ftmp -= (a3 - rhydrodir * cosf(t8)) * (-part / rhydromod);
		  	Ttmp += _cross<number, number4>(rhydrodir, a3) * part;

			Ttmp += _cross<number, number4>(ppos_base, Ftmp);

			Ftmp.w = hb_energy;
			F += Ftmp;
		}
	}
	// END HYDROGEN BONDING

	// CROSS STACKING
	number4 rcstack = rhydro;
	number rcstackmodsqr = rhydromodsqr;
	if(SQR(CRST_RCLOW) < rcstackmodsqr && rcstackmodsqr < SQR(CRST_RCHIGH)) {
	  	number rcstackmod = sqrtf(rcstackmodsqr);
	  	number4 rcstackdir = rcstack / rcstackmod;

		// angles involved in the CSTCK interaction
		number t1 = CUDA_LRACOS (-CUDA_DOT(a1, b1));
		number t2 = CUDA_LRACOS (-CUDA_DOT(b1, rcstackdir));
		number t3 = CUDA_LRACOS ( CUDA_DOT(a1, rcstackdir));
		number t4 = CUDA_LRACOS ( CUDA_DOT(a3, b3));
		number t7 = CUDA_LRACOS (-CUDA_DOT(rcstackdir, b3));
		number t8 = CUDA_LRACOS ( CUDA_DOT(rcstackdir, a3));

	 	 // functions called at their relevant arguments
		number f2 = _f2(rcstackmod, CRST_F2);
		number f4t1 = _f4(t1, CRST_THETA1_T0, CRST_THETA1_TS, CRST_THETA1_TC, CRST_THETA1_A, CRST_THETA1_B);
		number f4t2 = _f4(t2, CRST_THETA2_T0, CRST_THETA2_TS, CRST_THETA2_TC, CRST_THETA2_A, CRST_THETA2_B);
		number f4t3 = _f4(t3, CRST_THETA3_T0, CRST_THETA3_TS, CRST_THETA3_TC, CRST_THETA3_A, CRST_THETA3_B);
		number f4t4 = _f4(t4, CRST_THETA4_T0, CRST_THETA4_TS, CRST_THETA4_TC, CRST_THETA4_A, CRST_THETA4_B) +
				_f4(PI - t4, CRST_THETA4_T0, CRST_THETA4_TS, CRST_THETA4_TC, CRST_THETA4_A, CRST_THETA4_B);
		number f4t7 = _f4(t7, CRST_THETA7_T0, CRST_THETA7_TS, CRST_THETA7_TC, CRST_THETA7_A, CRST_THETA7_B) +
				_f4(PI - t7, CRST_THETA7_T0, CRST_THETA7_TS, CRST_THETA7_TC, CRST_THETA7_A, CRST_THETA7_B);
		number f4t8 = _f4(t8, CRST_THETA8_T0, CRST_THETA8_TS, CRST_THETA8_TC, CRST_THETA8_A, CRST_THETA8_B) +
				_f4(PI - t8, CRST_THETA8_T0, CRST_THETA8_TS, CRST_THETA8_TC, CRST_THETA8_A, CRST_THETA8_B);

		number cstk_energy = f2 * f4t1 * f4t2 * f4t3 * f4t4 * f4t7 * f4t8;

		if(cstk_energy < (number) 0) {
			// derivatives called at the relevant arguments
			number f2D = _f2D(rcstackmod, CRST_F2);
			number f4t1Dsin = -_f4Dsin(t1, CRST_THETA1_T0, CRST_THETA1_TS, CRST_THETA1_TC, CRST_THETA1_A, CRST_THETA1_B);
			number f4t2Dsin = -_f4Dsin(t2, CRST_THETA2_T0, CRST_THETA2_TS, CRST_THETA2_TC, CRST_THETA2_A, CRST_THETA2_B);
			number f4t3Dsin = _f4Dsin(t3, CRST_THETA3_T0, CRST_THETA3_TS, CRST_THETA3_TC, CRST_THETA3_A, CRST_THETA3_B);
			number f4t4Dsin = _f4Dsin(t4, CRST_THETA4_T0, CRST_THETA4_TS, CRST_THETA4_TC, CRST_THETA4_A, CRST_THETA4_B) -
					_f4Dsin(PI - t4, CRST_THETA4_T0, CRST_THETA4_TS, CRST_THETA4_TC, CRST_THETA4_A, CRST_THETA4_B);
			number f4t7Dsin = -_f4Dsin(t7, CRST_THETA7_T0, CRST_THETA7_TS, CRST_THETA7_TC, CRST_THETA7_A, CRST_THETA7_B) +
					_f4Dsin(PI - t7, CRST_THETA7_T0, CRST_THETA7_TS, CRST_THETA7_TC, CRST_THETA7_A, CRST_THETA7_B);
			number f4t8Dsin = _f4Dsin(t8, CRST_THETA8_T0, CRST_THETA8_TS, CRST_THETA8_TC, CRST_THETA8_A, CRST_THETA8_B) -
					_f4Dsin(PI - t8, CRST_THETA8_T0, CRST_THETA8_TS, CRST_THETA8_TC, CRST_THETA8_A, CRST_THETA8_B);

			// RADIAL PART
			Ftmp = rcstackdir * (cstk_energy * f2D / f2);

			// THETA1; t1 = LRACOS (-a1 * b1);
			Ttmp -= _cross<number, number4>(a1, b1) * (-cstk_energy * f4t1Dsin / f4t1);

			// TETA2; t2 = LRACOS (-b1 * rhydrodir);
			Ftmp -= (b1 + rcstackdir * cosf(t2)) * (cstk_energy * f4t2Dsin / (f4t2 * rcstackmod));

			// TETA3; t3 = LRACOS (a1 * rhydrodir);
			number part = -cstk_energy * f4t3Dsin / f4t3;
			Ftmp -= (a1 - rcstackdir * cosf(t3)) * (-part / rcstackmod);
			Ttmp += _cross<number, number4>(rcstackdir, a1) * part;

			// TETA4; t4 = LRACOS (a3 * b3);
			Ttmp -= _cross<number, number4>(a3, b3) * (-cstk_energy * f4t4Dsin / f4t4);

			// THETA7; t7 = LRACOS (-rcsrackir * b3);
			Ftmp -= (b3 + rcstackdir * cosf(t7)) * (cstk_energy * f4t7Dsin / (f4t7 * rcstackmod));

			// THETA 8; t8 = LRACOS (rhydrodir * a3);
			part = -cstk_energy * f4t8Dsin / f4t8;
			Ftmp -= (a3 - rcstackdir * cosf(t8)) * (-part / rcstackmod);
			Ttmp += _cross<number, number4>(rcstackdir, a3) * part;

			Ttmp += _cross<number, number4>(ppos_base, Ftmp);

			Ftmp.w = cstk_energy;
			F += Ftmp;
		}
	}

	// COAXIAL STACKING
	if (use_oxDNA2_coaxial_stacking){
		number4 rstack = r + qpos_stack - ppos_stack;
		number rstackmodsqr = CUDA_DOT(rstack, rstack);
		if(SQR(CXST_RCLOW) < rstackmodsqr && rstackmodsqr < SQR(CXST_RCHIGH)) {
			number rstackmod = sqrtf(rstackmodsqr);
			number4 rstackdir = rstack / rstackmod;

			// angles involved in the CXST interaction
			number t1 = CUDA_LRACOS (-CUDA_DOT(a1, b1));
			number t4 = CUDA_LRACOS ( CUDA_DOT(a3, b3));
			number t5 = CUDA_LRACOS ( CUDA_DOT(a3, rstackdir));
			number t6 = CUDA_LRACOS (-CUDA_DOT(b3, rstackdir));

			// functions called at their relevant arguments
			number f2 = _f2(rstackmod, CXST_F2);
			number f4t1 = _f4(t1, CXST_THETA1_T0_OXDNA2, CXST_THETA1_TS, CXST_THETA1_TC, CXST_THETA1_A, CXST_THETA1_B) +
				_f4_pure_harmonic(t1, CXST_THETA1_SA, CXST_THETA1_SB);
			number f4t4 = _f4(t4, CXST_THETA4_T0, CXST_THETA4_TS, CXST_THETA4_TC, CXST_THETA4_A, CXST_THETA4_B);
			number f4t5 = _f4(t5, CXST_THETA5_T0, CXST_THETA5_TS, CXST_THETA5_TC, CXST_THETA5_A, CXST_THETA5_B) +
					_f4(PI - t5, CXST_THETA5_T0, CXST_THETA5_TS, CXST_THETA5_TC, CXST_THETA5_A, CXST_THETA5_B);
			number f4t6 = _f4(t6, CXST_THETA6_T0, CXST_THETA6_TS, CXST_THETA6_TC, CXST_THETA6_A, CXST_THETA6_B) +
					_f4(PI - t6, CXST_THETA6_T0, CXST_THETA6_TS, CXST_THETA6_TC, CXST_THETA6_A, CXST_THETA6_B);

			number cxst_energy = f2 * f4t1 * f4t4 * f4t5 * f4t6;

			if(cxst_energy < (number) 0) {
				// derivatives called at the relevant arguments
				number f2D = _f2D(rstackmod, CXST_F2);
				number f4t1Dsin = -_f4Dsin(t1, CXST_THETA1_T0_OXDNA2, CXST_THETA1_TS, CXST_THETA1_TC, CXST_THETA1_A, CXST_THETA1_B) -
					_f4Dsin_pure_harmonic(t1, CXST_THETA1_SA, CXST_THETA1_SB);
				number f4t4Dsin =  _f4Dsin(t4, CXST_THETA4_T0, CXST_THETA4_TS, CXST_THETA4_TC, CXST_THETA4_A, CXST_THETA4_B);
				number f4t5Dsin =  _f4Dsin(t5, CXST_THETA5_T0, CXST_THETA5_TS, CXST_THETA5_TC, CXST_THETA5_A, CXST_THETA5_B) -
						_f4Dsin(PI - t5, CXST_THETA5_T0, CXST_THETA5_TS, CXST_THETA5_TC, CXST_THETA5_A, CXST_THETA5_B);
				number f4t6Dsin = -_f4Dsin(t6, CXST_THETA6_T0, CXST_THETA6_TS, CXST_THETA6_TC, CXST_THETA6_A, CXST_THETA6_B) +
						_f4Dsin(PI - t6, CXST_THETA6_T0, CXST_THETA6_TS, CXST_THETA6_TC, CXST_THETA6_A, CXST_THETA6_B);

				// RADIAL PART
				Ftmp = rstackdir * (cxst_energy * f2D / f2);

				// THETA1; t1 = LRACOS (-a1 * b1);
				Ttmp -= _cross<number, number4>(a1, b1) * (-cxst_energy * f4t1Dsin / f4t1);

				// TETA4; t4 = LRACOS (a3 * b3);
				Ttmp -= _cross<number, number4>(a3, b3) * (-cxst_energy * f4t4Dsin / f4t4);

				// THETA5; t5 = LRACOS ( a3 * rstackdir);
				number part = cxst_energy * f4t5Dsin / f4t5;
				Ftmp -= (a3 - rstackdir * cosf(t5)) / rstackmod * part;
				Ttmp -= _cross<number, number4>(rstackdir, a3) * part;

				// THETA6; t6 = LRACOS (-b3 * rstackdir);
				Ftmp -= (b3 + rstackdir * cosf(t6)) * (cxst_energy * f4t6Dsin / (f4t6 * rstackmod));

				Ttmp += _cross<number, number4>(ppos_stack, Ftmp);

				Ftmp.w = cxst_energy;
				F += Ftmp;
			}
		}
	}
	else {
		number4 rstack = r + qpos_stack - ppos_stack;
		number rstackmodsqr = CUDA_DOT(rstack, rstack);
		if(SQR(CXST_RCLOW) < rstackmodsqr && rstackmodsqr < SQR(CXST_RCHIGH)) {
			number rstackmod = sqrtf(rstackmodsqr);
			number4 rstackdir = rstack / rstackmod;

			// angles involved in the CXST interaction
			number t1 = CUDA_LRACOS (-CUDA_DOT(a1, b1));
			number t4 = CUDA_LRACOS ( CUDA_DOT(a3, b3));
			number t5 = CUDA_LRACOS ( CUDA_DOT(a3, rstackdir));
			number t6 = CUDA_LRACOS (-CUDA_DOT(b3, rstackdir));

			// This is the position the backbone would have with major-minor grooves the same width.
			// We need to do this to implement different major-minor groove widths because rback is
			// used as a reference point for things that have nothing to do with the actual backbone
			// position (in this case, the coaxial stacking interaction).
			number4 rbackboneref = r + POS_BACK * b1 - POS_BACK * a1;
			number rbackrefmod = _module<number, number4>(rbackboneref);
			number4 rbackbonerefdir = rbackboneref / rbackrefmod;
			number cosphi3 = CUDA_DOT(rstackdir, (_cross<number, number4>(rbackbonerefdir, a1)));

			// functions called at their relevant arguments
			number f2 = _f2(rstackmod, CXST_F2);
			number f4t1 = _f4(t1, CXST_THETA1_T0_OXDNA, CXST_THETA1_TS, CXST_THETA1_TC, CXST_THETA1_A, CXST_THETA1_B) +
					_f4(2 * PI - t1, CXST_THETA1_T0_OXDNA, CXST_THETA1_TS, CXST_THETA1_TC, CXST_THETA1_A, CXST_THETA1_B);
			number f4t4 = _f4(t4, CXST_THETA4_T0, CXST_THETA4_TS, CXST_THETA4_TC, CXST_THETA4_A, CXST_THETA4_B);
			number f4t5 = _f4(t5, CXST_THETA5_T0, CXST_THETA5_TS, CXST_THETA5_TC, CXST_THETA5_A, CXST_THETA5_B) +
					_f4(PI - t5, CXST_THETA5_T0, CXST_THETA5_TS, CXST_THETA5_TC, CXST_THETA5_A, CXST_THETA5_B);
			number f4t6 = _f4(t6, CXST_THETA6_T0, CXST_THETA6_TS, CXST_THETA6_TC, CXST_THETA6_A, CXST_THETA6_B) +
					_f4(PI - t6, CXST_THETA6_T0, CXST_THETA6_TS, CXST_THETA6_TC, CXST_THETA6_A, CXST_THETA6_B);
			number f5cosphi3 = _f5(cosphi3, CXST_F5_PHI3);

			number cxst_energy = f2 * f4t1 * f4t4 * f4t5 * f4t6 * SQR(f5cosphi3);

			if(cxst_energy < (number) 0) {
				// derivatives called at the relevant arguments
				number f2D = _f2D(rstackmod, CXST_F2);
				number f4t1Dsin = -_f4Dsin(t1, CXST_THETA1_T0_OXDNA, CXST_THETA1_TS, CXST_THETA1_TC, CXST_THETA1_A, CXST_THETA1_B) +
						_f4Dsin(2 * PI - t1, CXST_THETA1_T0_OXDNA, CXST_THETA1_TS, CXST_THETA1_TC, CXST_THETA1_A, CXST_THETA1_B);
				number f4t4Dsin =  _f4Dsin(t4, CXST_THETA4_T0, CXST_THETA4_TS, CXST_THETA4_TC, CXST_THETA4_A, CXST_THETA4_B);
				number f4t5Dsin =  _f4Dsin(t5, CXST_THETA5_T0, CXST_THETA5_TS, CXST_THETA5_TC, CXST_THETA5_A, CXST_THETA5_B) -
						_f4Dsin(PI - t5, CXST_THETA5_T0, CXST_THETA5_TS, CXST_THETA5_TC, CXST_THETA5_A, CXST_THETA5_B);
				number f4t6Dsin = -_f4Dsin(t6, CXST_THETA6_T0, CXST_THETA6_TS, CXST_THETA6_TC, CXST_THETA6_A, CXST_THETA6_B) +
						_f4Dsin(PI - t6, CXST_THETA6_T0, CXST_THETA6_TS, CXST_THETA6_TC, CXST_THETA6_A, CXST_THETA6_B);
				number f5cosphi3D = _f5D(cosphi3, CXST_F5_PHI3);

				// RADIAL PART
				Ftmp = rstackdir * (cxst_energy * f2D / f2);

				// THETA1; t1 = LRACOS (-a1 * b1);
				Ttmp -= _cross<number, number4>(a1, b1) * (-cxst_energy * f4t1Dsin / f4t1);

				// TETA4; t4 = LRACOS (a3 * b3);
				Ttmp -= _cross<number, number4>(a3, b3) * (-cxst_energy * f4t4Dsin / f4t4);

				// THETA5; t5 = LRACOS ( a3 * rstackdir);
				number part = cxst_energy * f4t5Dsin / f4t5;
				Ftmp -= (a3 - rstackdir * cosf(t5)) / rstackmod * part;
				Ttmp -= _cross<number, number4>(rstackdir, a3) * part;

				// THETA6; t6 = LRACOS (-b3 * rstackdir);
				Ftmp -= (b3 + rstackdir * cosf(t6)) * (cxst_energy * f4t6Dsin / (f4t6 * rstackmod));

				// COSPHI3
				number rbackrefmodcub = rbackrefmod * rbackrefmod * rbackrefmod;

				//number a1b1 = a1 * b1;
				number a2b1 = CUDA_DOT(a2, b1);
				number a3b1 = CUDA_DOT(a3, b1);
				number ra1 = CUDA_DOT(rstackdir, a1);
				number ra2 = CUDA_DOT(rstackdir, a2);
				number ra3 = CUDA_DOT(rstackdir, a3);
				number rb1 = CUDA_DOT(rstackdir, b1);

				number parentesi = (ra3 * a2b1 - ra2 * a3b1);
				number dcdr    = -GAMMA * parentesi * (GAMMA * (ra1 - rb1) + rstackmod) / rbackrefmodcub;
				number dcda1b1 =  GAMMA * SQR(GAMMA) * parentesi / rbackrefmodcub;
				number dcda2b1 =  GAMMA * ra3 / rbackrefmod;
				number dcda3b1 = -GAMMA * ra2 / rbackrefmod;
				number dcdra1  = -SQR(GAMMA) * parentesi * rstackmod / rbackrefmodcub;
				number dcdra2  = -GAMMA * a3b1 / rbackrefmod;
				number dcdra3  =  GAMMA * a2b1 / rbackrefmod;
				number dcdrb1  = -dcdra1;

				part = cxst_energy * 2 * f5cosphi3D / f5cosphi3;

				Ftmp -= part * (rstackdir * dcdr +
							    ((a1 - rstackdir * ra1) * dcdra1 +
								(a2 - rstackdir * ra2) * dcdra2 +
								(a3 - rstackdir * ra3) * dcdra3 +
								(b1 - rstackdir * rb1) * dcdrb1) / rstackmod);

				Ttmp += part * (_cross<number, number4>(rstackdir, a1) * dcdra1 +
							    _cross<number, number4>(rstackdir, a2) * dcdra2 +
							    _cross<number, number4>(rstackdir ,a3) * dcdra3);

				Ttmp -= part * (_cross<number, number4>(a1, b1) * dcda1b1 +
							    _cross<number, number4>(a2, b1) * dcda2b1 +
							    _cross<number, number4>(a3, b1) * dcda3b1);

				Ttmp += _cross<number, number4>(ppos_stack, Ftmp);

				Ftmp.w = cxst_energy;
				F += Ftmp;
			}
		}
	}
	
	// DEBYE HUCKEL
	if (use_debye_huckel){
		number rbackmod = _module<number, number4>(rbackbone);
		if (rbackmod < MD_dh_RC[0]){
			number4 rbackdir = rbackbone / rbackmod;
			if(rbackmod < MD_dh_RHIGH[0]){
				Ftmp = rbackdir * (-MD_dh_prefactor[0] * expf(MD_dh_minus_kappa[0] * rbackmod) * (MD_dh_minus_kappa[0] / rbackmod - 1.0f / SQR(rbackmod)));
			}
			else {
				Ftmp = rbackdir * (-2.0f * MD_dh_B[0] * (rbackmod - MD_dh_RC[0]));
			}

			// check for half-charge strand ends
			if (MD_dh_half_charged_ends[0] && (pbonds.n3 == P_INVALID || pbonds.n5 == P_INVALID)) {
				Ftmp *= 0.5f;
			}
			if (MD_dh_half_charged_ends[0] && (qbonds.n3 == P_INVALID || qbonds.n5 == P_INVALID)) {
				Ftmp *= 0.5f;
			}

			Ttmp -= _cross<number, number4>(ppos_back, Ftmp);
			F -= Ftmp;
		}
	}
	
	T += Ttmp;
	
	// this component stores the energy due to hydrogen bonding
	T.w = old_Tw + hb_energy;
}

template <typename number, typename number4>
__global__ void nucl_np_forces(number4 *poss, GPU_quat<number> *orientations,  number4 *forces, number4 *torques, int *matrix_neighs, int *number_neighs, LR_bonds *bonds, TLF_TMP xs) {
	if(IND >= MD_N[0]) return;

    const number4 ppos = poss[IND];
    if (get_particle_btype<number, number4>(ppos) == P_NP) return;

    number4 F          = forces[IND];
    number4 T          = make_number4<number, number4>(0, 0, 0, 0);
    const LR_bonds bs  = bonds[IND];
    const int n3       = bs.n3;
    const int n5       = bs.n5;
    const int n3_btype = get_particle_btype<number, number4>(poss[n3]);

    number4 a1; 
    number4 a2;
    number4 a3;
    get_vectors_from_quat<number,number4>(orientations[IND], a1, a2, a3); 
	
    if (n3 != P_INVALID) {
        const number4 qpos = poss[n3];
        number4 b1; 
        number4 b2;
        number4 b3;
        get_vectors_from_quat<number,number4>(orientations[n3], b1, b2, b3);

       
        if (n3_btype != P_NP) { 
            _bonded_part<number, number4, true>
                (ppos, a1, a2, a3, qpos, b1, b2, b3, F, T, xs.grooving, xs.use_oxDNA2_FENE);
        } else if (n3_btype == P_NP) { 
            number4 np_F = forces[n3];
            number4 np_T = make_number4<number, number4>(0, 0, 0, 0);
            _np_nucleotide_bonded<number, number4>
                (ppos, a1, a2, a3, qpos, b1, b2, b3, F, T, np_F, np_T);
            forces[n3]  = np_F;
            torques[n3] = np_T;
        } else {
            printf("something messed up!\n");
        }
    }

    if(n5 != P_INVALID) {
        const number4 qpos = poss[n5];
        number4 b1; 
        number4 b2;
        number4 b3;
        get_vectors_from_quat<number,number4>(orientations[n5], b1, b2, b3);
        _bonded_part<number, number4, false>
            (qpos, b1, b2, b3, ppos, a1, a2, a3, F, T, xs.grooving, xs.use_oxDNA2_FENE);
    }

	const int type = get_particle_type<number, number4>(ppos);
	const int num_neighs = number_neighs[IND];

	T.w = (number) 0;
	for(int j = 0; j < num_neighs; j++) {
		const int k_index  = matrix_neighs[j*MD_N[0] + IND];
        const number4 qpos = poss[k_index];
        const int k_btype  = get_particle_btype<number, number4>(qpos);
        if (k_btype != P_NP) {
            number4 b1; 
            number4 b2;
            number4 b3;
            get_vectors_from_quat<number,number4>(orientations[k_index], b1, b2, b3);
            // LR_bonds pbonds = bonds[IND]; // we grabbed this value earilier
            const LR_bonds qbonds = bonds[k_index];
            _particle_particle_interaction<number, number4>(ppos, a1, a2, a3, qpos, b1, b2, b3, F, T, xs.grooving, xs.use_debye_huckel, xs.use_oxDNA2_coaxial_stacking, bs, qbonds, IND, k_index);
        }
	}
	
	T = _vectors_transpose_number4_product(a1, a2, a3, T);

	// the real energy per particle is half of the one computed (because we count each interaction twice)
	F.w *= (number) 0.5f;
	T.w *= (number) 0.5f;
	forces[IND] = F;
	torques[IND] = T;

}

template <typename number, typename number4>
__global__ void nucl_np_forces_no_list(number4 *poss, GPU_quat<number> *orientations, number4 *forces, number4 *torques, LR_bonds *bonds, TLF_TMP xs) {
	if(IND >= MD_N[0]) return;

    const number4 ppos = poss[IND];
    if (get_particle_btype<number, number4>(ppos) == P_NP) return;

    number4 F          = forces[IND];
    number4 T          = make_number4<number, number4>(0, 0, 0, 0);
    const LR_bonds bs  = bonds[IND];
    const int n3       = bs.n3;
    const int n5       = bs.n5;
    const int n3_btype = get_particle_btype<number, number4>(poss[n3]);

    number4 a1; 
    number4 a2;
    number4 a3;
    get_vectors_from_quat<number,number4>(orientations[IND], a1, a2, a3); 
    
    if (n3 != P_INVALID) {
        const number4 qpos = poss[n3];
        number4 b1; 
        number4 b2;
        number4 b3;
        get_vectors_from_quat<number,number4>(orientations[n3], b1, b2, b3);

       
        if (n3_btype != P_NP) { 
            _bonded_part<number, number4, true>
                (ppos, a1, a2, a3, qpos, b1, b2, b3, F, T, xs.grooving, xs.use_oxDNA2_FENE);
        } else if (n3_btype == P_NP) { 
            number4 np_F = forces[n3];
            number4 np_T = make_number4<number, number4>(0, 0, 0, 0);
            _np_nucleotide_bonded<number, number4>
                (ppos, a1, a2, a3, qpos, b1, b2, b3, F, T, np_F, np_T);
            forces[n3]  = np_F;
            torques[n3] = np_T;
        } else {
            printf("something messed up!\n");
        }
    }

    if(n5 != P_INVALID) {
        const number4 qpos = poss[n5];
        number4 b1; 
        number4 b2;
        number4 b3;
        get_vectors_from_quat<number,number4>(orientations[n5], b1, b2, b3);
        _bonded_part<number, number4, false>
            (qpos, b1, b2, b3, ppos, a1, a2, a3, F, T, xs.grooving, xs.use_oxDNA2_FENE);
    }

    T.w = (number) 0;
    for(int j = 0; j < MD_N[0]; j++) {
        const int j_btype = get_particle_btype<number, number4>(poss[j]);
        if (j != IND && n3 != j && n5 != j && j_btype != P_NP) {
            const number4 qpos = poss[j];
            number4 b1; 
            number4 b2;
            number4 b3;
            get_vectors_from_quat<number,number4>(orientations[j], b1, b2, b3);
            const LR_bonds qbonds = bonds[j];
            _particle_particle_interaction<number, number4>
                (ppos, a1, a2, a3, qpos, b1, b2, b3, F, T, true, true, true, bs, qbonds, IND, j);
        }
    }

    T = _vectors_transpose_number4_product(a1, a2, a3, T);

    F.w *= (number) 0.5f;
    T.w *= (number) 0.5f;

    forces[IND]  = F;
    torques[IND] = T;
}


template<typename number, typename number4>
void CUDANBLOCKInteraction<number, number4>::compute_forces(CUDABaseList<number, number4> *lists, number4 *d_poss, GPU_quat<number> *d_orientations, number4 *d_forces, number4 *d_torques, LR_bonds *d_bonds) {
    /*
	CUDASimpleVerletList<number, number4> *_v_lists = dynamic_cast<CUDASimpleVerletList<number, number4> *>(lists);
        TLF: For base implementation I've removed the following block:
                if (_v_lists)
                    if (_v_lists use_edge()) 
                        dna_forces_edge_nonbonded
                        dna_forces_edge_bonded
                    else
                        dna_forces (with verlet edge list)

             We can add these methods back later after base implementation is complete.
             See CUDADNAInteraction<...>::compute_forces(...) for more details.
    */

	CUDASimpleVerletList<number, number4> *_v_lists = dynamic_cast<CUDASimpleVerletList<number, number4> *>(lists);
	if(_v_lists != NULL) {
        nucl_np_forces<number, number4>
            <<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
            (d_poss, d_orientations, d_forces, d_torques, _v_lists->_d_matrix_neighs, _v_lists->_d_number_neighs, d_bonds, tmp); // TLF
            //(d_poss, d_orientations, d_forces, d_torques, _v_lists->_d_matrix_neighs, _v_lists->_d_number_neighs, d_bonds, this->_grooving, _use_debye_huckel, _use_oxDNA2_coaxial_stacking, _use_oxDNA2_FENE);
        CUT_CHECK_ERROR("forces_second_step simple_lists error");
    } else {
		nucl_np_forces_no_list<number, number4>
			<<<this->_launch_cfg.blocks, this->_launch_cfg.threads_per_block>>>
			    (d_poss, d_orientations,  d_forces, d_torques, d_bonds, tmp); // TLF
        CUT_CHECK_ERROR("forces_second_step no_lists error");
    }
    
}

template class CUDANBLOCKInteraction<float, float4>;
template class CUDANBLOCKInteraction<double, LR_double4>;
